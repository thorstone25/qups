#include "hip/hip_runtime.h"
// # include "hip/hip_vector_types.h" // vector math

// # include "sizes.cu" // size defines

# include "interpd.cu" // samplers using constant sizing

// # include "half2_math.h" // vector math for half types only 
template<typename T2, typename U, typename U3>
__device__ void greens_temp(T2 * __restrict__ y, 
    const U * __restrict__ Pi, const T2 * __restrict__ a, 
    const U * __restrict__ Pr, const U * __restrict__ Pv, 
    const T2 * __restrict__ x, const U * __restrict__ sb, const size_t * iblock,
	const U s0t0fscinv[6],
    const int * E, const int iflag
    ) {

    // extract time parameters
    const U s0   = s0t0fscinv[0];
    const U t0   = s0t0fscinv[1];
    const U fs   = s0t0fscinv[2];
    const U fsr  = s0t0fscinv[3];
    const U cinv = s0t0fscinv[4];
    const U R0   = s0t0fscinv[5]; // min distance for div by 0
    
    // get starting index of this scatterer
    const int s = threadIdx.x + blockIdx.x * blockDim.x; // time 
    const int n = threadIdx.y + blockIdx.y * blockDim.y; // rx
    const int m = threadIdx.z + blockIdx.z * blockDim.z; // tx

    // reinterpret inputs as vector pointers (makes loading faster and indexing easier)
    const U3 * pi = reinterpret_cast<const U3*>(Pi); // 3 x I
    const U3 * pr = reinterpret_cast<const U3*>(Pr); // 3 x N x E
    const U3 * pv = reinterpret_cast<const U3*>(Pv); // 3 x M x E

    // rename for readability
    const int N = QUPS_N, S = QUPS_S, M = QUPS_M, T = QUPS_T;
    const size_t I = QUPS_I;
    // rxs, num scat, output time size, txs, kernel time size, 
    // S is size of output, T is size of input kernel, I the number of scats
    
    // temp vars
    // const float ts = s + s0*fs; // compute the time index for this thread
    const T2 zero_v = {0, 0}; // OOB value, power scaling
    // const U A0 = (R0) ? 1 / (R0 * R0) : 1; // max power scaling
    U r1, r2, tau; // length, time (tmp values)
    T2 val = zero_v; // accumulator

    // if valid scat, for each tx/rx
    if(s < S){
        for(size_t i = iblock[2*blockIdx.x+0]; i <= iblock[2*blockIdx.x+1] && i < I; ++i){ // for each scatterer
            if(s >= sb[2*i+0]){ // if within sampling window
                # pragma unroll 
                for(int me = 0; me < E[1]; ++me){ // for each tx sub-aperture
                    # pragma unroll 
                    for(int ne = 0; ne < E[0]; ++ne){ // for each rx sub-aperture
    
                        // 2-way path distance
                        r1 = length(pi[i] - pr[n + ne*N]); // rx propagation
                        r2 = length(pi[i] - pv[m + me*M]); // tx propagation
                        
                        // get kernel delay for the scatterer
                        tau = (U)s - (cinv * (r1 + r2) + t0 - s0)*fs;

                        // limit to minimum dist of R0
                        if(R0){
                            r1 /= R0; r1 = (r1 < 1) ? 1 : r1;
                            r2 /= R0; r2 = (r2 < 1) ? 1 : r2;
                        } else {
                            r1 = 1;
                            r2 = 1;
                        }
                        
                        // sample the kernel and add to the signal at this time
                        // fsr applies a 'stretch' operation to the sample time, because the 
                        // input data x is sampled at sampling frequency fsr * fs
                        val += a[i] * sample(x, fsr*tau, iflag, zero_v, T) / (r1 * r2); // out of bounds: extrap 0
                    }
                }
            }
        }
        
        // output signal when all scatterers and sub-apertures are sampled
        // normalize by the discrete length of the signal, and the upsampling factor
        y[s + n*S + m*N*S] =  val / (R0*R0*fsr);
    }
}

__global__ void greensf(float2 * __restrict__ y, 
    const float * __restrict__ Pi, const float2 * __restrict__ a, 
    const float * __restrict__ Pr, const float * __restrict__ Pv, 
    const float2 * __restrict__ x, const float * __restrict__ sb, const size_t * iblock,
	const float s0t0fscinv[6],
    const int * E, const int iflag
    ) {
    greens_temp<float2, float, float3>(y, Pi, a, Pr, Pv, x, sb, iblock, s0t0fscinv, E, iflag);
}

__global__ void greens(double2 * __restrict__ y, 
    const double * __restrict__ Pi, const double2 * __restrict__ a, 
    const double * __restrict__ Pr, const double * __restrict__ Pv, 
    const double2 * __restrict__ x, const double * __restrict__ sb, const size_t * iblock,
	const double s0t0fscinv[6],
    const int * E, const int iflag
    ) {
    greens_temp<double2, double, double3>(y, Pi, a, Pr, Pv, x, sb, iblock, s0t0fscinv, E, iflag);
}

#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half2 operator/(half2 a, float b){
    return make_half2((float)a.x / b, (float)a.y / b); // scale amplitude in FP32 precision
}

__global__ void greensh(ushort2 * __restrict__ y, 
    const float * __restrict__ Pi, const short2 * __restrict__ a, 
    const float * __restrict__ Pr, const float * __restrict__ Pv, 
    const ushort2 * __restrict__ x, const float * __restrict__ sb,const size_t * iblock,
	const float s0t0fscinv[6],
    const int * E, const int iflag
    ) {
    greens_temp<half2, float, float3>((half2 *)y, Pi, (const half2 *)a, Pr, Pv, (const half2 *)x, sb, iblock, s0t0fscinv, E, iflag);
}
#endif
