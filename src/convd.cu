
#include <hip/hip_runtime.h>
#if (__CUDA_ARCH__ >= 530)
#define __CUDA_NO_HALF2_OPERATORS__ // block half2 vector math operators
#include <cuda_fp16.h> // define half/half2 types, without half2 operators
#endif

// real/complex conjugation
inline __host__ __device__ float conj(const float a) {
    return a; 
}
inline __host__ __device__ double conj(const double a) {
    return a;
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half conj(const half a) {
    return a;
}
#endif

inline __host__ __device__ float2 conj(const float2 a) {
    return make_float2(a.x, -a.y); 
}
inline __host__ __device__ double2 conj(const double2 a) {
    return make_double2(a.x, -a.y); 
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half2 conj(const half2 a) {
    return make_half2(a.x, -a.y); 
}
#endif


// complex multiplication
inline __host__ __device__ float2 operator*(const float2 a, const float2 b) {
    return make_float2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
inline __host__ __device__ double2 operator*(const double2 a, const double2 b) {
    return make_double2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half2 operator*(const half2 a, const half2 b) {
    return make_half2(a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x);
}
#endif

inline __host__ __device__ float2 operator*(const float2 a, const float b){
    return make_float2(b*a.x, b*a.y);
}
inline __host__ __device__ double2 operator*(const double2 a, const double b){
    return make_double2(b*a.x, b*a.y);
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ half2 operator*(const half2 a, const half b){
    return make_half2(b*a.x, b*a.y);
}
#endif

// real/complex addition/assignment
inline __host__ __device__ void operator+=(float2 &a, const float2 b){
    a.x += b.x;
    a.y += b.y;
}
inline __host__ __device__ void operator+=(double2 &a, const double2 b){
    a.x += b.x;
    a.y += b.y;
}
#if (__CUDA_ARCH__ >= 530)
inline __host__ __device__ void operator+=(half2 &a, const half2 b){
    a.x += b.x;
    a.y += b.y;
}
#endif


/*
* Compute the cross correlation of two sets of data. The data will be
* correlated in the first dimension. M >= N must be satisfied.
*
* Inputs:
*  x:         first signal  (M x S)
*  y:         second signal (N x S)
*
* Outputs:
*  z:         resulting cross correlation
*
*
*/


# ifndef L0
__constant__ int L0; // starting lag
# endif

// xcorr template
template <typename T>
inline __device__ void conv_temp(const T * const x, const T * const y, T * __restrict__ z, T za, size_t * sizes){
    /*    xcorr_temp(const T* x, const T* y, T* z, T za)
     x, y: input array pointer(s)
     za: 0 value for the data type
     z:    output array pointer
     cross correlation 
    */

    // get stride and len
    const int istr  = sizes[0];
    const int ilen  = sizes[1];
    const int jstr  = sizes[2];
    const int jlen  = sizes[3];
    const int lstr  = sizes[4];
    const int llen  = sizes[5];
    const int clen  = sizes[6];
    
    const int I = ilen * clen;
    const int J = jlen * clen;
    const int L = llen * clen;
    
    const int c = threadIdx.x + blockDim.x*blockIdx.x; // column output index 
    const int l = threadIdx.y + blockDim.y*blockIdx.y; // strided output index 
    const int s = blockIdx.z; // batch index

    // if valid lag indices, multiply and accumulate in-place
    if(l < L && c < clen)
        # pragma unroll
        for(int i = 0, j = L0 - l; i < ilen || j < jlen; ++i, ++j)
            if(0 <= i && i < ilen && 0 <= j && j < jlen) // signal in bounds
                za += x[c + i*istr + s*I] * y[c + (jlen - 1 - j)*jstr + s*J]; // accum the cross product

    // output result:
    if(l < L && c < clen)
        z[c + l*lstr + s*L] = za;
}

// xcorr kernels
__global__ void convf(const float* x, const float* y, float* __restrict__ z, size_t * sizes){
    conv_temp<float>(x, y, z, 0.0f, sizes);
}

__global__ void conv(const double* x, const double* y, double* __restrict__ z, size_t * sizes){
    conv_temp<double>(x, y, z, 0.0, sizes);
}
#if (__CUDA_ARCH__ >= 530)
__global__ void convh(const unsigned short* x, const unsigned short* y, unsigned short* __restrict__ z, size_t * sizes){
    conv_temp<half>((half*)x, (half*)y, (half*)z, 0.0f, sizes);
}
#endif
__global__ void convcf(const float2* x, const float2* y, float2* __restrict__ z, size_t * sizes){
    conv_temp<float2>(x, y, z, make_float2(0.0f,0.0f), sizes);
}

__global__ void convc(const double2* x, const double2* y, double2* __restrict__ z, size_t * sizes){
    conv_temp<double2>(x, y, z, make_double2(0.0,0.0), sizes);
}
#if (__CUDA_ARCH__ >= 530)
__global__ void convch(const ushort2* x, const ushort2* y, ushort2* __restrict__ z, size_t * sizes){
    conv_temp<half2>((half2*)x, (half2*)y, (half2*)z, make_half2(0.0f, 0.0f), sizes);
}
#endif


